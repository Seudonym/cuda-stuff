#include "hip/hip_runtime.h"
#include "io/config.hpp"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <cstdint>

// Add error checking macro
#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)

__global__ void init_curand_states(hiprandState *states, unsigned long seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &states[id]);
}

__global__ void buddhabrot_kernel(uint32_t *histogram, RenderConfig config,
                                  hiprandState *states, volatile int *progress)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state = states[thread_id];

    float x_range = config.x_max - config.x_min;
    float y_range = config.y_max - config.y_min;

    for (size_t sample = 0; sample < config.samples_per_thread; ++sample)
    {
        float cr = hiprand_uniform(&state) * x_range + config.x_min;
        float ci = hiprand_uniform(&state) * y_range + config.y_min;
        float zr = 0.0f, zi = 0.0f;
        float temp_zr = 0.0f;

        size_t iteration = 0, iterations_to_escape = 0;
        bool escaped = false;

        for (iteration = 0; iteration < config.max_iterations; ++iteration)
        {
            temp_zr = zr;
            zr = zr * zr - zi * zi + cr;
            zi = 2.0f * temp_zr * zi + ci;
            if (zr * zr + zi * zi > 8.0f)
            {
                escaped = true;
                iterations_to_escape = iteration;
                break;
            }
        }

        if (!escaped)
            continue;

        zr = 0.0f, zi = 0.0f;
        for (iteration = 0; iteration < iterations_to_escape; ++iteration)
        {
            temp_zr = zr;
            zr = zr * zr - zi * zi + cr;
            zi = 2.0f * temp_zr * zi + ci;

            int pixel_x = static_cast<int>(((zr - config.x_min) / x_range) * config.width);
            int pixel_y = static_cast<int>(((zi - config.y_min) / y_range) * config.height);

            if (pixel_x >= 0 && pixel_x < static_cast<int>(config.width) &&
                pixel_y >= 0 && pixel_y < static_cast<int>(config.height))
            {
                atomicAdd(&histogram[pixel_y * config.width + pixel_x], 1);
            }
        }
    }

    // Report progress after this thread completes all samples
    // Only one thread per block reports to reduce atomic contention
    if (threadIdx.x == 0)
    {
        atomicAdd((int *)progress, 1);
        __threadfence_system();
    }

    states[thread_id] = state;
}

__global__ void buddhabrot_rgb_kernel(uint32_t *r_hist, uint32_t *g_hist, uint32_t *b_hist, RenderConfig config,
                                      hiprandState *states, volatile int *progress)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state = states[thread_id];

    float x_range = config.x_max - config.x_min;
    float y_range = config.y_max - config.y_min;

    for (size_t sample = 0; sample < config.samples_per_thread; ++sample)
    {
        float cr = hiprand_uniform(&state) * x_range + config.x_min;
        float ci = hiprand_uniform(&state) * y_range + config.y_min;
        float zr = 0.0f, zi = 0.0f;
        float temp_zr = 0.0f;

        size_t iteration = 0, iterations_to_escape = 0;
        bool escaped = false;

        for (iteration = 0; iteration < config.max_iterations; ++iteration)
        {
            temp_zr = zr;
            zr = zr * zr - zi * zi + cr;
            zi = 2.0f * temp_zr * zi + ci;
            iterations_to_escape++;
            if (zr * zr + zi * zi > 4.0f)
            {
                escaped = true;
                break;
            }
        }

        if (!escaped)
            continue;

        zr = 0.0f, zi = 0.0f;
        for (iteration = 0; iteration < iterations_to_escape; ++iteration)
        {
            temp_zr = zr;
            zr = zr * zr - zi * zi + cr;
            zi = 2.0f * temp_zr * zi + ci;

            int pixel_x = static_cast<int>(((zr - config.x_min) / x_range) * config.width);
            int pixel_y = static_cast<int>(((zi - config.y_min) / y_range) * config.height);

            if (pixel_x >= 0 && pixel_x < static_cast<int>(config.width) &&
                pixel_y >= 0 && pixel_y < static_cast<int>(config.height))
            {
                if (iterations_to_escape < config.b_thresh)
                    atomicAdd(&b_hist[pixel_y * config.width + pixel_x], 1);
                if (iterations_to_escape < config.g_thresh)
                    atomicAdd(&g_hist[pixel_y * config.width + pixel_x], 1);
                if (iterations_to_escape < config.r_thresh)
                    atomicAdd(&r_hist[pixel_y * config.width + pixel_x], 1);
            }
        }
    }

    // Report progress after this thread completes all samples
    // Only one thread per block reports to reduce atomic contention
    if (threadIdx.x == 0)
    {
        atomicAdd((int *)progress, 1);
        __threadfence_system();
    }

    states[thread_id] = state;
}

void launch_buddhabrot_kernel(uint32_t *histogram, RenderConfig config)
{
    dim3 threads_per_block(256);
    dim3 num_blocks(1024);

    // Allocate mapped pinned memory for progress tracking
    volatile int *h_progress, *d_progress;
    hipSetDeviceFlags(hipDeviceMapHost);
    cudaCheckErrors("hipSetDeviceFlags error");
    hipHostAlloc((void **)&h_progress, sizeof(int), hipHostMallocMapped);
    cudaCheckErrors("hipHostAlloc error");
    hipHostGetDevicePointer((int **)&d_progress, (int *)h_progress, 0);
    cudaCheckErrors("hipHostGetDevicePointer error");
    *h_progress = 0;

    hiprandState *d_states;
    hipMalloc(&d_states, threads_per_block.x * num_blocks.x * sizeof(hiprandState));
    cudaCheckErrors("hipMalloc states error");
    init_curand_states<<<num_blocks, threads_per_block>>>(d_states, time(NULL));
    hipDeviceSynchronize();
    cudaCheckErrors("init hiprand error");

    uint32_t *d_histogram;
    hipMalloc(&d_histogram, config.width * config.height * sizeof(uint32_t));
    cudaCheckErrors("hipMalloc histogram error");
    hipMemset(d_histogram, 0, config.width * config.height * sizeof(uint32_t));
    cudaCheckErrors("hipMemset error");

    printf("Rendering buddhabrot...\n");

    // Launch kernel with progress tracking
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    buddhabrot_kernel<<<num_blocks, threads_per_block>>>(d_histogram, config, d_states, d_progress);
    cudaCheckErrors("Kernel launch error");

    hipEventRecord(stop);

    // Poll for progress updates
    unsigned int total_blocks = num_blocks.x * num_blocks.y;
    int last_value = 0;
    float last_progress = 0.0f;

    printf("Progress:\n");
    do
    {
        hipEventQuery(stop); // May help on Windows WDDM driver model
        int current_value = *h_progress;
        float current_progress = (float)current_value / (float)total_blocks;

        if ((current_progress - last_progress) >= 0.02f)
        {
            printf("  %2.0f%% complete (%d/%d blocks)\n",
                   current_progress * 100.0f, current_value, total_blocks);
            last_progress = current_progress;
        }
        last_value = current_value;
    } while (last_progress < 0.98f);
    printf("\n");

    // Wait for completion and measure time
    hipEventSynchronize(stop);
    cudaCheckErrors("event sync error");

    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    printf("Rendering complete! Elapsed time: %.2f ms (%.2f seconds)\n",
           elapsed_ms, elapsed_ms / 1000.0f);

    // Copy results back
    hipMemcpy(histogram, d_histogram, config.width * config.height * sizeof(uint32_t),
               hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy histogram error");

    // Cleanup
    hipFree(d_histogram);
    hipFree(d_states);
    hipHostFree((void *)h_progress);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void launch_buddhabrot_rgb_kernel(uint32_t *r_hist, uint32_t *g_hist, uint32_t *b_hist, RenderConfig config)
{
    dim3 threads_per_block(256);
    dim3 num_blocks(1024);

    // Allocate mapped pinned memory for progress tracking
    volatile int *h_progress, *d_progress;
    hipSetDeviceFlags(hipDeviceMapHost);
    cudaCheckErrors("hipSetDeviceFlags error");
    hipHostAlloc((void **)&h_progress, sizeof(int), hipHostMallocMapped);
    cudaCheckErrors("hipHostAlloc error");
    hipHostGetDevicePointer((int **)&d_progress, (int *)h_progress, 0);
    cudaCheckErrors("hipHostGetDevicePointer error");
    *h_progress = 0;

    hiprandState *d_states;
    hipMalloc(&d_states, threads_per_block.x * num_blocks.x * sizeof(hiprandState));
    cudaCheckErrors("hipMalloc states error");
    init_curand_states<<<num_blocks, threads_per_block>>>(d_states, time(NULL));
    hipDeviceSynchronize();
    cudaCheckErrors("init hiprand error");

    uint32_t *d_r_hist, *d_g_hist, *d_b_hist;
    hipMalloc(&d_r_hist, config.width * config.height * sizeof(uint32_t));
    hipMalloc(&d_g_hist, config.width * config.height * sizeof(uint32_t));
    hipMalloc(&d_b_hist, config.width * config.height * sizeof(uint32_t));
    cudaCheckErrors("hipMalloc histogram error");
    hipMemset(d_r_hist, 0, config.width * config.height * sizeof(uint32_t));
    hipMemset(d_g_hist, 0, config.width * config.height * sizeof(uint32_t));
    hipMemset(d_b_hist, 0, config.width * config.height * sizeof(uint32_t));
    cudaCheckErrors("hipMemset error");

    printf("Rendering buddhabrot...\n");

    // Launch kernel with progress tracking
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    buddhabrot_rgb_kernel<<<num_blocks, threads_per_block>>>(d_r_hist, d_g_hist, d_b_hist, config, d_states, d_progress);
    cudaCheckErrors("Kernel launch error");

    hipEventRecord(stop);

    // Poll for progress updates
    unsigned int total_blocks = num_blocks.x * num_blocks.y;
    int last_value = 0;
    float last_progress = 0.0f;

    printf("Progress:\n");
    do
    {
        hipEventQuery(stop); // May help on Windows WDDM driver model
        int current_value = *h_progress;
        float current_progress = (float)current_value / (float)total_blocks;

        if ((current_progress - last_progress) >= 0.02f)
        {
            printf("  %2.0f%% complete (%d/%d blocks)\n",
                   current_progress * 100.0f, current_value, total_blocks);
            last_progress = current_progress;
        }
        last_value = current_value;
    } while (last_progress <= 0.98f);
    printf("\n");

    // Wait for completion and measure time
    hipEventSynchronize(stop);
    cudaCheckErrors("event sync error");

    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    printf("Rendering complete! Elapsed time: %.2f ms (%.2f seconds)\n",
           elapsed_ms, elapsed_ms / 1000.0f);

    // Copy results back
    hipMemcpy(r_hist, d_r_hist, config.width * config.height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(g_hist, d_g_hist, config.width * config.height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(b_hist, d_b_hist, config.width * config.height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy histogram error");

    // Cleanup
    hipFree(d_r_hist);
    hipFree(d_g_hist);
    hipFree(d_b_hist);
    hipFree(d_states);
    hipHostFree((void *)h_progress);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}