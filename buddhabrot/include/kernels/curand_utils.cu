
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init_curand_states(hiprandState *states, unsigned long seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &states[id]);
}